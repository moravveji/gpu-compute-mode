#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

void print_gpu_info(void) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);

    fprintf(stdout, "maxBlocksPerMultiProcessor=%d, \
            multiProcessorCount=%d, \
            maxGridSize[0]=%d, \
            maxThreadsDim[0]=%d, \
            maxThreadsPerBlock=%d,\n", \
            dev.maxBlocksPerMultiProcessor, \
            dev.multiProcessorCount, \
            dev.maxGridSize[0], \
            dev.maxThreadsDim[0], \
            dev.maxThreadsPerBlock);
}

int get_maxThreadsPerBlock(void) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    return dev.maxThreadsPerBlock;
}

int get_num_gpus(void) {
    int ndev;
    hipGetDeviceCount(&ndev);
    return ndev;
}

int get_num_blocks(int nranks, int nomp) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    int nblocks = (dev.maxBlocksPerMultiProcessor * dev.multiProcessorCount) / (nranks * nomp);

    return nblocks;
}

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

float call_saxpy(int narr, int nblocks, int nthreads) {
    float *x, *y, *dx, *dy;  // host and device x and y arrays
    x = (float*) malloc(narr * sizeof(float));
    y = (float*) malloc(narr * sizeof(float));

    hipMalloc(&dx, narr * sizeof(float));
    hipMalloc(&dy, narr * sizeof(float));

    for (int i=0; i<narr; i++) {
        x[i] = 1.0f; y[i] = 2.0f;
    }

    hipMemcpy(dx, x, narr*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, narr*sizeof(float), hipMemcpyHostToDevice);

    // int nblocks = (narr + nthreads - 1) / nthreads;
    saxpy<<<nblocks, nthreads>>>(narr, 2.0f, dx, dy);

    hipMemcpy(y, dy, narr*sizeof(float), hipMemcpyDeviceToHost);

    float maxerr = 0.0f;
    for (int i=0; i<narr; i++) {
        maxerr = max(maxerr, abs(y[i] - 4.0f));
    }

    hipFree(dx); hipFree(dy);
    free(x); free(y);

    return maxerr;
}
