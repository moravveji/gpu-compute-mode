

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void print_gpu_info(void) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);

    fprintf(stdout, "maxBlocksPerMultiProcessor=%d, \
            multiProcessorCount=%d, \
            maxGridSize[0]=%d, \
            maxThreadsDim[0]=%d, \
            maxThreadsPerBlock=%d,\n", \
            dev.maxBlocksPerMultiProcessor, \
            dev.multiProcessorCount, \
            dev.maxGridSize[0], \
            dev.maxThreadsDim[0], \
            dev.maxThreadsPerBlock);
}

int get_maxThreadsPerBlock(void) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    return dev.maxThreadsPerBlock;
}

int get_num_gpus(void) {
    int ndev;
    hipGetDeviceCount(&ndev);
    return ndev;
}

int get_num_blocks(int nranks, int nomp) {
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    int nblocks = (dev.maxBlocksPerMultiProcessor * dev.multiProcessorCount) / (nranks * nomp);

    return nblocks;
}

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

float call_saxpy(int narr, int nblocks, int nthreads) {
    float *x, *y, *dx, *dy;  // host and device x and y arrays
    x = (float*) malloc(narr * sizeof(float));
    y = (float*) malloc(narr * sizeof(float));
    if (x == NULL || y == NULL) {
        fprintf(stderr, "ERROR: malloc failed");
        exit(21);
    }

    if (hipMalloc((void**) &dx, narr * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc failed for dx");
        exit(21);
    }
    if (hipMalloc((void**) &dy, narr * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc failed for dy");
        exit(21);
    }

    for (int i=0; i<narr; i++) {
        x[i] = 1.0f; y[i] = 2.0f;
    }

    hipMemcpy(dx, x, narr*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, narr*sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<nblocks, nthreads>>>(narr, 2.0f, dx, dy);

    hipMemcpy(y, dy, narr*sizeof(float), hipMemcpyDeviceToHost);

    float maxerr = 0.0f;
    for (int i=0; i<narr; i++) {
        float locerr = fabs(y[i] - 4.0f);
        maxerr = (locerr > maxerr) ? locerr : maxerr;
    }

    hipFree(dx); hipFree(dy);
    free(x); free(y);

    return maxerr;
}
