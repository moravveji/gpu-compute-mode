#include "hip/hip_runtime.h"

#include <stdlib.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

float call_saxpy(int narr, int nthreads) {
    float *x, *y, *dx, *dy;  // host and device x and y arrays
    x = (float*) malloc(narr * sizeof(float));
    y = (float*) malloc(narr * sizeof(float));

    hipMalloc(&dx, narr * sizeof(float));
    hipMalloc(&dy, narr * sizeof(float));

    for (int i=0; i<narr; i++) {
        x[i] = 1.0f; y[i] = 2.0f;
    }

    hipMemcpy(dx, x, narr*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, narr*sizeof(float), hipMemcpyHostToDevice);

    int nblocks = (narr + nthreads - 1) / nthreads;
    saxpy<<<nblocks, nthreads>>>(narr, 2.0f, dx, dy);

    hipMemcpy(y, dy, narr*sizeof(float), hipMemcpyDeviceToHost);

    float maxerr = 0.0f;
    for (int i=0; i<narr; i++) {
        maxerr = max(maxerr, abs(y[i] - 4.0f));
    }

    hipFree(dx); hipFree(dy);
    free(x); free(y);

    return maxerr;
}
